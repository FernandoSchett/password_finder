/*
@(#)File:           bruteforce_cuda.cu
@(#)Last changed:   Date: 11/24/2022 21:40:00
@(#)Purpose:        Brute Force Password Breaker with GPU  
@(#)Modified:       Fernando Schettini, Gabriel Mascarenhas, Jadson Nobre
@(#)Usage:
 (*) Hotocompile:   nvcc bruteforce_cuda.cu -o bruteforce_cuda
 (*) Hotoexecute:   ./bruteforce <password> <numbers_of_blocks>
                    ./bruteforce senhatez 30
@(#)Comment:
 (*)   Disponible in: http://forum.guiadohacker.com.br/showthread.php?t=40571
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h> 
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h> // importa a biblioteca CUDA
#include <string>
#include <cstring>
#include <assert.h>
using namespace std; //define o namespace padrão

//97 to 122 use only lowercase letters
//65 to 90 use only capital letters
//48 to 57 use only numbers
#define START_CHAR 48
#define END_CHAR 122
#define MAXIMUM_PASSWORD 20

__device__ long long my_pow(long long x, int y){//funcao para calcular potencia, marcada com __device__ para ser executada no device 

  long long res = 1;

  if (y == 0)
    return res;
  else
    return x * my_pow(x, y-1);

}
__device__ int my_strlen(char *s) {
    int sum = 0;
    while (*s++) sum++;
    return sum;
 }

__global__ void bruteForce(char *pass) { //define a função que poderá ser chamada globalmente, como kernel 
  long long j = blockIdx.x*blockDim.x+threadIdx.x;
  int tam = my_strlen(pass);
  int pass_b26[MAXIMUM_PASSWORD];

  long long int pass_decimal = 0;
  int base = END_CHAR-START_CHAR+2;

  for(int i = 0; i < tam; i++){
    pass_b26[i] = (int)pass[i]-START_CHAR+1;
  }
  for(int i = tam - 1; i > -1; i--){
    pass_decimal += (long long int) pass_b26[i]*my_pow(base,i);
  }

  long long int max = my_pow(base,tam);
  char s[MAXIMUM_PASSWORD];

  while(j < max){
      if(j == pass_decimal){
        printf("Encontrou o password!\n");
        int index = 0;
        printf("O número que estamos tentando encontrar (password na base decimal): %lli\n", j);
        while(j > 0){
          s[index++] = START_CHAR + j%base-1;
          j /= base;
        }
        s[index] = '\0';
        printf("Password encontrado: %s\n", s);
        assert(0);
        break;
      }
      j += blockDim.x*gridDim.x; //move o stride loop
  }
    
}

void printTime(char *text, double time){
  FILE *f = fopen("cuda_time.txt", "a");
  fprintf(f, "%s %1.2f\n", text, time);
  fclose(f);
}

int main(int argc, char **argv) {
  int id;
  int numsms;
  hipGetDevice(&id); //pega o identificador do device
  hipDeviceGetAttribute(&numsms, hipDeviceAttributeMultiprocessorCount, id); //pega o número de multiprocessadores do device e o número de SMS
  char *password_gpu; //ponteiro para a variável que será alocada no device
  
  hipMallocManaged(&password_gpu, MAXIMUM_PASSWORD); //linka ás memórias do device e do host a variável password_gpu
  strcpy(password_gpu, argv[1]); // copia a variavel para duas memorias

  time_t t1, t2;
  double dif;
  int blocos = numsms * atoi(argv[2]); //define o número de blocos
  time(&t1);
  bruteForce<<< blocos, 1024 >>>(password_gpu); //chama o kernel, passando o numero de blocos e threads por bloco
  hipDeviceSynchronize(); // Espera todas as threads terminarem dentro da GPU
  time(&t2);
  dif = difftime (t2,t1);
  printf("\n%1.2f seconds\n", dif);
  printTime(argv[1], dif);
  hipFree(password_gpu);
  return 0;

}